#include "hip/hip_runtime.h"

//#include "hip/hip_runtime.h"
//#include ""
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <stdio.h>
#include <iostream>
#include <time.h>

using namespace cv;
using namespace std;


#define CHECK(value) {                                          \
    hipError_t _m_cudaStat = value;                                        \
    if (_m_cudaStat != hipSuccess) {                                       \
        cout<< "Error:" << hipGetErrorString(_m_cudaStat) \
            << " at line " << __LINE__ << " in file " << __FILE__ << "\n"; \
        exit(1);                                                            \
    } }


__global__ void mirror(unsigned char* input, unsigned char* output,
                            int num_of_pixels, int num_of_rows,
                            int num_of_cols)
{
	const int i = 3*(blockIdx.x * blockDim.x + threadIdx.x);
	if(i >= num_of_pixels) return;
	const int rowi = i/(3*num_of_cols);
	const int offset = i - rowi*(3*num_of_cols);
	const int mirror_i = rowi*(3*num_of_cols) + 3*num_of_cols - offset;
	if (i <= rowi*(3*num_of_cols) + 3*num_of_cols/2){
		output[i] = input[mirror_i];
		output[i + 1] = input[mirror_i + 1];
		output[i + 2] = input[mirror_i + 2];
		output[mirror_i] = input[i];
		output[mirror_i + 1] = input[i + 1];
		output[mirror_i + 2] = input[i + 2];
	}
}

int main( int argc, char** argv )
{
	/************************************************/

    Mat image1,image2;
    image1 = imread("cat4.jpg", CV_LOAD_IMAGE_COLOR);
	image2 = imread("cat4.jpg", CV_LOAD_IMAGE_COLOR);

    if(! image1.data && !image2.data)
    {
        cout <<  "Could not open or find the image" << std::endl ;
        return -1;
    }
    namedWindow( "Display window INPUT", WINDOW_AUTOSIZE );
    imshow("Display window INPUT",image1);
	hipEvent_t startCUDA, stopCUDA;
    clock_t startCPU;
    float elapsedTimeCUDA, elapsedTimeCPU;
	hipEventCreate(&startCUDA);
    hipEventCreate(&stopCUDA);

	/************************************************/

	startCPU = clock();
	for(int i = 0; i < image1.rows; i++)
    {
        Vec3b* p = image1.ptr<Vec3b>(i);
        for (int j = 0, reverse_j = image1.cols - 1; j < reverse_j; j++, reverse_j-- ){
            for (int ch = 0; ch < 3; ch++)
                swap(p[j][ch],p[reverse_j][ch]);
        }
    }
	elapsedTimeCPU = (double)(clock()-startCPU)/CLOCKS_PER_SEC;
    cout << "CPU time = " << elapsedTimeCPU*1000 << " ms\n";

	/************************************************/

    unsigned char *d_input,*d_output;
    int num_of_pixels = 3*image2.rows*image2.cols;
    CHECK(hipMalloc(&d_input, num_of_pixels));
	CHECK(hipMalloc(&d_output, num_of_pixels));
    CHECK(hipMemcpy(d_input, image2.data, num_of_pixels, hipMemcpyHostToDevice));
	hipEventRecord(startCUDA,0);
    mirror<<<(num_of_pixels/3 + 255)/256,256>>>(d_input,d_output,num_of_pixels,image2.rows,image2.cols);
	hipEventRecord(stopCUDA,0);
    hipEventSynchronize(stopCUDA);
    CHECK(hipGetLastError());
	hipEventElapsedTime(&elapsedTimeCUDA, startCUDA, stopCUDA);
    cout << "CUDA time = " << elapsedTimeCUDA << " ms\n";

	/************************************************/

    CHECK(hipMemcpy(image2.data, d_output, num_of_pixels, hipMemcpyDeviceToHost));
    CHECK(hipFree(d_input));
	CHECK(hipFree(d_output));
	imwrite("outputCPU.jpg",image1);
    imwrite("outputCUDA.jpg",image2);
	//namedWindow( "Display window CPU", WINDOW_AUTOSIZE );
    namedWindow( "Display window CUDA", WINDOW_AUTOSIZE  );
	//imshow("Display window CPU", image1);
	imshow("Display window CUDA", image2);
    waitKey(0);
    return 0;
}
